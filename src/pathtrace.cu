#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cfloat>
#include <algorithm>
#include <limits>
#include <vector>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <fstream>
#include <chrono>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

// ─────────── BVH DATA STRUCTURES ───────────
struct AABB {
    glm::vec3 min, max;
};

struct BVHNodeGPU {
    AABB bounds;
    int left, right;    // child indices, -1 for leaf
    int geomIndex;      // leaf: index into geoms array
};

AABB computeBounds(const Geom& g) {
    glm::vec3 corners[8] = {
        {-0.5f,-0.5f,-0.5f},{+0.5f,-0.5f,-0.5f},
        {-0.5f,+0.5f,-0.5f},{+0.5f,+0.5f,-0.5f},
        {-0.5f,-0.5f,+0.5f},{+0.5f,-0.5f,+0.5f},
        {-0.5f,+0.5f,+0.5f},{+0.5f,+0.5f,+0.5f}
    };
    AABB box;
    box.min = glm::vec3(std::numeric_limits<float>::max());
    box.max = glm::vec3(-std::numeric_limits<float>::max());
    for (int i = 0; i < 8; ++i) {
        glm::vec4 w = g.transform * glm::vec4(corners[i], 1.0f);
        box.min = glm::min(box.min, glm::vec3(w));
        box.max = glm::max(box.max, glm::vec3(w));
    }
    return box;
}

int buildBVHRecursive(
    const std::vector<AABB>& bboxes,
    std::vector<int>& indices,
    int start, int end,
    std::vector<BVHNodeGPU>& nodes)
{
    int nodeIdx = (int)nodes.size();
    nodes.push_back({});
    int count = end - start;
    if (count == 1) {
        nodes[nodeIdx].bounds    = bboxes[indices[start]];
        nodes[nodeIdx].left      = -1;
        nodes[nodeIdx].right     = -1;
        nodes[nodeIdx].geomIndex = indices[start];
        return nodeIdx;
    }
    // centroid bbox
    AABB cbox;
    cbox.min = glm::vec3(std::numeric_limits<float>::max());
    cbox.max = glm::vec3(-std::numeric_limits<float>::max());
    for (int i = start; i < end; ++i) {
        const AABB &b = bboxes[indices[i]];
        glm::vec3 cent = (b.min + b.max) * 0.5f;
        cbox.min = glm::min(cbox.min, cent);
        cbox.max = glm::max(cbox.max, cent);
    }
    glm::vec3 extent = cbox.max - cbox.min;
    int axis = (extent.x > extent.y && extent.x > extent.z) ? 0
             : (extent.y > extent.z) ? 1 : 2;
    std::sort(indices.begin() + start, indices.begin() + end,
        [&](int a, int b) {
            const AABB &ba = bboxes[a], &bb = bboxes[b];
            float ca = (ba.min[axis] + ba.max[axis]) * 0.5f;
            float cb = (bb.min[axis] + bb.max[axis]) * 0.5f;
            return ca < cb;
        });
    int mid = start + count/2;
    int leftChild  = buildBVHRecursive(bboxes, indices, start, mid, nodes);
    int rightChild = buildBVHRecursive(bboxes, indices, mid, end, nodes);
    nodes[nodeIdx].left      = leftChild;
    nodes[nodeIdx].right     = rightChild;
    nodes[nodeIdx].geomIndex = -1;
    // union bounds
    const AABB &bl = nodes[leftChild].bounds;
    const AABB &br = nodes[rightChild].bounds;
    nodes[nodeIdx].bounds.min = glm::min(bl.min, br.min);
    nodes[nodeIdx].bounds.max = glm::max(bl.max, br.max);
    return nodeIdx;
}

void buildBVH(const std::vector<Geom>& geoms, std::vector<BVHNodeGPU>& nodes) {
    int n = (int)geoms.size();
    std::vector<AABB> bboxes(n);
    for (int i = 0; i < n; ++i)
        bboxes[i] = computeBounds(geoms[i]);
    std::vector<int> indices(n);
    for (int i = 0; i < n; ++i) indices[i] = i;
    nodes.clear();
    buildBVHRecursive(bboxes, indices, 0, n, nodes);
}

__device__ bool intersectAABB(const AABB &box, const Ray &r) {
    float tmin = 0.0f, tmax = FLT_MAX;
    for (int i = 0; i < 3; ++i) {
        float invD = 1.0f / r.direction[i];
        float t0   = (box.min[i] - r.origin[i]) * invD;
        float t1   = (box.max[i] - r.origin[i]) * invD;
        if (invD < 0.0f) {
            // manual swap (std::swap not allowed in device code)
            float tmp = t0; t0 = t1; t1 = tmp;
        }
        tmin = fmaxf(tmin, t0);
        tmax = fminf(tmax, t1);
        if (tmax <= tmin) return false;
    }
    return true;
}

// ─────────── GLOBAL BVH STORAGE ───────────
static BVHNodeGPU* dev_bvhNodes = nullptr;
static int          h_bvhNodeCount = 0;

// ─────────── Timing & Error Checking ───────────
hipEvent_t startKernel, stopKernel;
float totalKernelTime = 0.0f;
#define ERRORCHECK 1
#define MAX_MATERIALS 64
#define FILENAME (strrchr(__FILE__,'/')?strrchr(__FILE__,'/')+1:__FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) return;
    fprintf(stderr,"CUDA error (%s:%d): %s: %s\n",
            file, line, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
#endif
}

// Group3 Mod - PSNR reference frame storage
static bool firstFrame = true;
static std::vector<glm::vec3> referenceFrame;

// Group3 Mod - Performance metrics
struct PerformanceMetrics {
    float totalRenderTime   = 0.0f;
    float avgIterationTime  = 0.0f;
    float samplesPerSecond  = 0.0f;
    size_t gpuMemoryUsed    = 0;
    float lastPSNR          = 0.0f;
    int   iterationsToClean = -1;
    std::chrono::high_resolution_clock::time_point startTime;
    void start() {
        startTime = std::chrono::high_resolution_clock::now();
    }
    void end(int iter, int pixelcount) {
        auto endTime = std::chrono::high_resolution_clock::now();
        float secs = std::chrono::duration<float>(endTime - startTime).count();
        totalRenderTime  += secs;
        avgIterationTime  = (totalRenderTime / iter) * 1000.0f;
        samplesPerSecond  = (pixelcount * float(iter)) / totalRenderTime;
    }
};
static PerformanceMetrics metrics;

void updateGpuMemory() {
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    metrics.gpuMemoryUsed = totalMem - freeMem;
}

float computePSNR(const std::vector<glm::vec3>& currentRaw, int iter) {
    std::vector<glm::vec3> current = currentRaw;
    for (auto& c : current) c /= float(iter);
    if (firstFrame && iter == 10) {
        referenceFrame = current;
        firstFrame = false;
        return FLT_MAX;
    }
    if (firstFrame) return FLT_MAX;
    double mse = 0.0;
    for (size_t i = 0; i < current.size(); ++i) {
        glm::vec3 d = current[i] - referenceFrame[i];
        mse += glm::dot(d, d);
    }
    mse /= (current.size() * 3.0);
    if (mse <= 1e-12) return FLT_MAX;
    return 10.0f * log10f(1.0f / float(mse));
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1<<31) | (depth<<22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

// Group3 Mod - Helper functions for improved lighting and reflections
__device__ glm::vec3 sampleHemisphere(float u1, float u2) {
    float r = sqrt(1.0f - u1 * u1);
    float phi = 2.0f * M_PI * u2;
    return glm::vec3(r * cos(phi), u1, r * sin(phi));
}

__device__ void createLocalCoordinateSystem(const glm::vec3& normal, glm::vec3& tangent, glm::vec3& bitangent) {
    if (fabs(normal.x) > fabs(normal.y)) {
        tangent = glm::normalize(glm::vec3(normal.z, 0, -normal.x));
    } else {
        tangent = glm::normalize(glm::vec3(0, -normal.z, normal.y));
    }
    bitangent = glm::cross(normal, tangent);
}

__device__ glm::vec3 sampleCosineWeightedHemisphere(float u1, float u2, const glm::vec3& normal) {
    glm::vec3 tangent, bitangent;
    createLocalCoordinateSystem(normal, tangent, bitangent);
    
    // Cosine-weighted hemisphere sampling
    float theta = acos(sqrt(1.0f - u1));
    float phi = 2.0f * M_PI * u2;
    
    float x = sin(theta) * cos(phi);
    float y = cos(theta);
    float z = sin(theta) * sin(phi);
    
    return glm::normalize(tangent * x + normal * y + bitangent * z);
}

__device__ glm::vec3 reflect(const glm::vec3& incident, const glm::vec3& normal) {
    return incident - 2.0f * glm::dot(incident, normal) * normal;
}

__device__ float schlickFresnel(float cosTheta, float n1, float n2) {
    float r0 = (n1 - n2) / (n1 + n2);
    r0 *= r0;
    return r0 + (1.0f - r0) * pow(1.0f - cosTheta, 5.0f);
}

__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
                               int iter, glm::vec3* image)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < resolution.x && y < resolution.y) {
        int idx = x + y*resolution.x;
        glm::vec3 pix = image[idx] / float(iter);
        pix = glm::pow(pix, glm::vec3(1.0f/2.2f));
        glm::ivec3 col;
        col.x = glm::clamp(int(pix.x*255.0f), 0, 255);
        col.y = glm::clamp(int(pix.y*255.0f), 0, 255);
        col.z = glm::clamp(int(pix.z*255.0f), 0, 255);
        pbo[idx].w = 0;
        pbo[idx].x = col.x;
        pbo[idx].y = col.y;
        pbo[idx].z = col.z;
    }
}

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < cam.resolution.x && y < cam.resolution.y) {
        int idx = x + y*cam.resolution.x;
        PathSegment &seg = pathSegments[idx];
        seg.ray.origin = cam.position;
        seg.color = glm::vec3(1.0f);
        seg.ray.direction = glm::normalize(
            cam.view
            - cam.right * cam.pixelLength.x * (float(x) - cam.resolution.x*0.5f)
            - cam.up    * cam.pixelLength.y * (float(y) - cam.resolution.y*0.5f)
        );
        seg.pixelIndex = idx;
        seg.remainingBounces = traceDepth;
    }
}

__global__ void computeIntersections(
    int depth, int num_paths,
    PathSegment* pathSegments,
    Geom* geoms, int geoms_size,
    ShadeableIntersection* intersections,
    BVHNodeGPU* bvhNodes)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    Ray ray = pathSegments[idx].ray;
    float t_min = FLT_MAX;
    int   hitG  = -1;

    int stack[64], sp = 0;
    stack[sp++] = 0; // root node

    while (sp > 0) {
        BVHNodeGPU node = bvhNodes[stack[--sp]];
        if (!intersectAABB(node.bounds, ray)) continue;
        if (node.left < 0) {
            int g = node.geomIndex;
            glm::vec3 pt, nrm; bool out;
            float t = (geoms[g].type == CUBE)
                ? boxIntersectionTest(geoms[g], ray, pt, nrm, out)
                : sphereIntersectionTest(geoms[g], ray, pt, nrm, out);
            if (t > 0 && t < t_min) {
                t_min = t; hitG = g;
                intersections[idx].point         = pt;
                intersections[idx].surfaceNormal = nrm;
                intersections[idx].outsideObject = out;  // Group3 Mod - Track whether ray hit from outside
            }
        } else {
            stack[sp++] = node.left;
            stack[sp++] = node.right;
        }
    }

    if (hitG < 0) {
        intersections[idx].t = -1.0f;
    } else {
        intersections[idx].t          = t_min;
        intersections[idx].materialId = geoms[hitG].materialid;
        intersections[idx].geomIndex  = hitG;  // Group3 Mod - Store geometry index
    }
}

// Group3 Mod - Improved physically-based shading kernel
__global__ void shadeAndExtendRays(
    int iter, int depth, int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials, int materialCount,
    glm::vec3* lightPositions, int numLights)
{
    extern __shared__ Material sharedMat[];
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    // load materials into shared memory
    for (int i = threadIdx.x; i < materialCount && i < MAX_MATERIALS; i += blockDim.x)
        sharedMat[i] = materials[i];
    __syncthreads();

    const ShadeableIntersection hit = shadeableIntersections[idx];
    PathSegment &segment = pathSegments[idx];
    
    // For missed rays or depleted bounce count
    if (hit.t < 0.0f || segment.remainingBounces <= 0) {
        // Group3 Mod - Environment lighting
        if (hit.t < 0.0f) {
            // Simple sky/environment light contribution
            float t = 0.5f * (segment.ray.direction.y + 1.0f);
            glm::vec3 skyColor = (1.0f - t) * glm::vec3(1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
            segment.color *= skyColor * 0.5f;  // Dimmer sky for better contrast
        }
        segment.remainingBounces = 0;
        return;
    }
    
    auto rng = makeSeededRandomEngine(iter, idx, depth);
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
    
    Material material = sharedMat[hit.materialId];
    
    // Emissive surfaces (lights)
    if (material.emittance > 0.0f) {
        segment.color *= material.color * material.emittance;
        segment.remainingBounces = 0;  // terminate path at light sources
        return;
    }

    // Group3 Mod - Russian roulette path termination
    if (depth > 3) {  // Start Russian Roulette after a few bounces
        float continueProbability = fmaxf(material.color.x, fmaxf(material.color.y, material.color.z));
        if (u01(rng) > continueProbability) {
            segment.remainingBounces = 0;
            return;
        }
        segment.color /= continueProbability;  // Compensate for termination probability
    }

    glm::vec3 hitPoint = hit.point;
    glm::vec3 normal = hit.surfaceNormal;
    glm::vec3 viewDir = -segment.ray.direction;  // Direction toward camera
    
    // Group3 Mod - Material-based shading
    segment.remainingBounces--;
    
    // Choose between specular reflection and diffuse based on material properties
    // Using hasReflective as reflectivity strength and hasRefractive (inverted) as roughness
    float reflectivity = material.hasReflective;
    float roughness = 1.0f - material.hasRefractive;
    
    if (reflectivity > 0.0f && u01(rng) < reflectivity) {
        // Specular reflection (mirror-like)
        glm::vec3 reflectDir = reflect(segment.ray.direction, normal);
        
        // Add some roughness/perturbation if needed
        if (roughness > 0.0f) {
            glm::vec3 tangent, bitangent;
            createLocalCoordinateSystem(reflectDir, tangent, bitangent);
            float angle = roughness * u01(rng) * M_PI * 0.5f;
            float x = sin(angle) * cos(2.0f * M_PI * u01(rng));
            float y = cos(angle);
            float z = sin(angle) * sin(2.0f * M_PI * u01(rng));
            reflectDir = glm::normalize(tangent * x + reflectDir * y + bitangent * z);
        }
        
        // Set up next ray
        segment.ray.origin = hitPoint + normal * 0.001f;  // Offset to avoid self-intersection
        segment.ray.direction = reflectDir;
        
        // For pure reflection, maintain color but apply material color tint
        segment.color *= material.specular.color;  // Use specular color for reflections
    } else {
        // Diffuse reflection (Lambertian)
        float u1 = u01(rng);
        float u2 = u01(rng);
        
        glm::vec3 diffuseDir = sampleCosineWeightedHemisphere(u1, u2, normal);
        
        // Set up next ray
        segment.ray.origin = hitPoint + normal * 0.001f;
        segment.ray.direction = diffuseDir;
        
        // Apply material color for diffuse reflection
        segment.color *= material.color;
    }
}

__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* paths) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < nPaths) {
        image[paths[idx].pixelIndex] += paths[idx].color;
    }
}

static Scene*                    hst_scene      = nullptr;
static GuiDataContainer*         guiData        = nullptr;
static glm::vec3*                dev_image      = nullptr;
static Geom*                     dev_geoms      = nullptr;
static Material*                 dev_materials  = nullptr;
static PathSegment*              dev_paths      = nullptr;
static ShadeableIntersection*    dev_intersections = nullptr;

// Group3 Mod - Light positions for direct lighting
static glm::vec3*                dev_lightPositions = nullptr;
static int                       h_numLights = 0;

void InitDataContainer(GuiDataContainer* imGuiData) {
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
    hst_scene = scene;
    const Camera &cam = scene->state.camera;
    int pixelcount   = cam.resolution.x * cam.resolution.y;

    // image & paths
    hipMalloc(&dev_image,  pixelcount*sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount*sizeof(glm::vec3));
    hipMalloc(&dev_paths,  pixelcount*sizeof(PathSegment));

    // geoms
    int G = (int)scene->geoms.size();
    hipMalloc(&dev_geoms,    G*sizeof(Geom));
    hipMemcpy(dev_geoms,     scene->geoms.data(), G*sizeof(Geom), hipMemcpyHostToDevice);

    // materials
    int M = (int)scene->materials.size();
    hipMalloc(&dev_materials, M*sizeof(Material));
    hipMemcpy(dev_materials,  scene->materials.data(), M*sizeof(Material), hipMemcpyHostToDevice);

    // build & upload BVH
    {
        std::vector<BVHNodeGPU> h_bvh;
        buildBVH(scene->geoms, h_bvh);
        h_bvhNodeCount = (int)h_bvh.size();
        hipMalloc(&dev_bvhNodes, h_bvhNodeCount*sizeof(BVHNodeGPU));
        hipMemcpy(dev_bvhNodes, h_bvh.data(), h_bvhNodeCount*sizeof(BVHNodeGPU), hipMemcpyHostToDevice);
    }

    // Group3 Mod - Find light sources for direct lighting
    std::vector<glm::vec3> lightPositions;
    for (size_t i = 0; i < scene->geoms.size(); ++i) {
        if (scene->materials[scene->geoms[i].materialid].emittance > 0.0f) {
            // Extract center of geometry as light position
            glm::vec4 center = scene->geoms[i].transform * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f);
            lightPositions.push_back(glm::vec3(center));
        }
    }
    h_numLights = (int)lightPositions.size();
    if (h_numLights > 0) {
        hipMalloc(&dev_lightPositions, h_numLights * sizeof(glm::vec3));
        hipMemcpy(dev_lightPositions, lightPositions.data(), h_numLights * sizeof(glm::vec3), hipMemcpyHostToDevice);
    }

    // intersections buffer
    hipMalloc(&dev_intersections, pixelcount*sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount*sizeof(ShadeableIntersection));

    // timing events
    hipEventCreate(&startKernel);
    hipEventCreate(&stopKernel);

    updateGpuMemory();
    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_bvhNodes);
    if (dev_lightPositions) hipFree(dev_lightPositions);  // Group3 Mod
    checkCUDAError("pathtraceFree");
}

void pathtrace(uchar4* pbo, int frame, int iter) {
    const Camera &cam = hst_scene->state.camera;
    int pixelcount   = cam.resolution.x * cam.resolution.y;

    dim3 blockSize2d(8,8),
         blocks2d((cam.resolution.x+7)/8, (cam.resolution.y+7)/8);
    int blockSize1d = 128;

    metrics.start();

    float rayGenTime=0, intersectTime=0, shadeTime=0, gatherTime=0;
    float totalK = 0.0f;

    // Ray generation
    hipEventRecord(startKernel);
    generateRayFromCamera<<<blocks2d,blockSize2d>>>(cam, iter, hst_scene->state.traceDepth, dev_paths);
    hipEventRecord(stopKernel);
    hipEventSynchronize(stopKernel);
    hipEventElapsedTime(&rayGenTime, startKernel, stopKernel);
    totalK += rayGenTime;
    checkCUDAError("generate camera ray");

    // Intersection & shading loop
    int depth = 0;
    int num_paths = pixelcount;
    bool iterationComplete = false;

    // Group3 Mod - Limit the number of path segments by active rays only
    int* dev_numActiveRays;
    hipMalloc(&dev_numActiveRays, sizeof(int));
    hipMemcpy(dev_numActiveRays, &pixelcount, sizeof(int), hipMemcpyHostToDevice);

    while (!iterationComplete && depth < hst_scene->state.traceDepth) {
        hipMemset(dev_intersections, 0, pixelcount*sizeof(ShadeableIntersection));
        int numBlocks1d = (num_paths + blockSize1d - 1) / blockSize1d;

        // BVH-based intersection
        hipEventRecord(startKernel);
        computeIntersections<<<numBlocks1d,blockSize1d>>>(
            depth, num_paths,
            dev_paths,
            dev_geoms, (int)hst_scene->geoms.size(),
            dev_intersections,
            dev_bvhNodes
        );
        hipEventRecord(stopKernel);
        hipEventSynchronize(stopKernel);
        hipEventElapsedTime(&intersectTime, startKernel, stopKernel);
        totalK += intersectTime;
        checkCUDAError("trace one bounce");

        // Group3 Mod - Physically based shading
        hipEventRecord(startKernel);
        int matCount = (int)hst_scene->materials.size();
        size_t shMemBytes = matCount * sizeof(Material);
        shadeAndExtendRays<<<numBlocks1d,blockSize1d,shMemBytes>>>(
            iter, depth, num_paths,
            dev_intersections,
            dev_paths,
            dev_materials, matCount,
            dev_lightPositions, h_numLights
        );
        hipEventRecord(stopKernel);
        hipEventSynchronize(stopKernel);
        hipEventElapsedTime(&shadeTime, startKernel, stopKernel);
        totalK += shadeTime;
        checkCUDAError("shade");

        depth++;
        
        // Group3 Mod - Exit condition based on maximum depth reached
        if (depth >= hst_scene->state.traceDepth) {
            iterationComplete = true;
        }
    }
    
    hipFree(dev_numActiveRays);  // Group3 Mod - Cleanup

    // Final gather
    int numBlocksPix = (pixelcount + blockSize1d - 1) / blockSize1d;
    hipEventRecord(startKernel);
    finalGather<<<numBlocksPix,blockSize1d>>>(pixelcount, dev_image, dev_paths);
    hipEventRecord(stopKernel);
    hipEventSynchronize(stopKernel);
    hipEventElapsedTime(&gatherTime, startKernel, stopKernel);
    totalK += gatherTime;
    checkCUDAError("finalGather");

    // Display
    sendImageToPBO<<<blocks2d,blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
    checkCUDAError("sendImageToPBO");

    // PSNR & metrics
    std::vector<glm::vec3> current(pixelcount);
    hipMemcpy(current.data(), dev_image, pixelcount*sizeof(glm::vec3), hipMemcpyDeviceToHost);

    metrics.end(iter, pixelcount);
    updateGpuMemory();
    float psnr = computePSNR(current, iter);
    metrics.lastPSNR = psnr;
    if (psnr > 35.0f && metrics.iterationsToClean < 0)
        metrics.iterationsToClean = iter;

    printf("\n====== PERFORMANCE METRICS SUMMARY ======\n");
    printf("Total render time: %.2f seconds\n", metrics.totalRenderTime);
    printf("Avg iteration time: %.2f ms\n",    metrics.avgIterationTime);
    printf("Samples per second: %.2f million rays/s\n", metrics.samplesPerSecond/1e6f);
    printf("GPU memory used: %.2f MB\n", metrics.gpuMemoryUsed / float(1<<20));
    if (psnr == FLT_MAX)    printf("PSNR: Inf dB\n");
    else                    printf("PSNR: %.2f dB\n", psnr);
    if (metrics.iterationsToClean > 0)
        printf("Iterations to clean: %d\n", metrics.iterationsToClean);
    printf("Total kernel time: %.2f ms\n", totalK);
    printf("  - Ray generation:   %.2f ms\n", rayGenTime);
    printf("  - Intersection:     %.2f ms\n", intersectTime);
    printf("  - Shading:          %.2f ms\n", shadeTime);
    printf("  - Final gather:     %.2f ms\n", gatherTime);
    printf("=========================================\n");

    hipMemcpy(hst_scene->state.image.data(),
               dev_image,
               pixelcount*sizeof(glm::vec3),
               hipMemcpyDeviceToHost);
    checkCUDAError("pathtrace");
}
